#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "texture.h"
#include "FreeImage.h"
#include ""

__device__ __inline__ uchar4
to_uchar4(float4 vec) {
    return make_uchar4(
            (unsigned char)vec.x,
            (unsigned char)vec.y,
            (unsigned char)vec.w,
            (unsigned char)vec.z);
}

__device__ __inline__ float4
operator+(const float4& lhs, const float4& rhs) {
    return make_float4(
            lhs.x + rhs.x,
            lhs.y + rhs.y,
            lhs.z + rhs.z,
            lhs.w + rhs.w);
}

__device__ __inline__ float4
operator/(const float4& lhs, const float& rhs) {
    return make_float4(lhs.x / rhs, lhs.y / rhs, lhs.z / rhs, lhs.w / rhs);
}

__global__ void textureDownsampling(
        hipSurfaceObject_t mipmap_output,
        hipTextureObject_t mipmap_input,
        uint32_t width,
        uint32_t height
) {
    uint32_t pixelX = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t pixelY = threadIdx.y + blockIdx.y * blockDim.y;
    if ( pixelX >= width || pixelY >= height ) return;

    float inv_w = 1.0f / float(width);
    float inv_h = 1.0f / float(height);

    float4 rgba =
            tex2D<float4>(mipmap_input, inv_w * (pixelX + 0), inv_h * (pixelY + 0)) +
            tex2D<float4>(mipmap_input, inv_w * (pixelX + 1), inv_h * (pixelY + 0)) +
            tex2D<float4>(mipmap_input, inv_w * (pixelX + 1), inv_h * (pixelY + 1)) +
            tex2D<float4>(mipmap_input, inv_w * (pixelX + 0), inv_h * (pixelY + 1));
    rgba = rgba / 4.0f;

    surf2Dwrite(rgba, mipmap_output, pixelX * sizeof(float4), pixelY);
}

Texture::Texture(const std::string &image_path): base_image(image_path) {

    hipError_t error;
    m_width = base_image.m_width;
    m_height = base_image.m_height;

    hipResourceDesc resource_desc = {};
    auto* base_data = base_image.image.data();

    auto num_levels = static_cast<uint32_t>(std::log2(std::max(base_image.m_width, base_image.m_height)));

    hipExtent extent = make_hipExtent(m_width, m_height, 0);
    m_channel_format = hipCreateChannelDesc<float4>();
    error = hipMallocMipmappedArray(&m_d_mipmap_array, &m_channel_format, extent, num_levels);
    if ( error != hipSuccess ) {
        std::cerr << "Error: Failed to allocate mipmap array with code " << error << std::endl;
    }

    // Copy base image to device
    hipArray_t base;
    error = hipGetMipmappedArrayLevel(&base, m_d_mipmap_array, 0);
    if ( error != hipSuccess ) {
        std::cerr << "Error: Failed to get mipmap array with code " << error << std::endl;
    }

    hipMemcpy3DParms copy_params = {0};
    copy_params.srcPtr          = make_hipPitchedPtr(base_data, m_width * sizeof(float4), m_width, m_height);
    copy_params.dstArray        = base;
    copy_params.extent.width    = m_width;
    copy_params.extent.height   = m_height;
    copy_params.extent.depth    = 1;
    copy_params.kind            = hipMemcpyHostToDevice;

    error = hipMemcpy3D(&copy_params);
    if ( error != hipSuccess ) {
        std::cerr << "Error: Failed to copy base mipmap to mipmap array with code " << error << std::endl;
    }

    for ( uint32_t level = 1; level < num_levels; ++level ) {
        hipArray_t level_from;
        hipArray_t level_to;

        // Get current level of mipmap
        error = hipGetMipmappedArrayLevel(&level_to, m_d_mipmap_array, level);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to get current level mipmap with code " << error << std::endl;
        }

        // Get last level of mipmap
        error = hipGetMipmappedArrayLevel(&level_from, m_d_mipmap_array, level -1);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to get last level mipmap with code " << error << std::endl;
        }

        hipExtent level_to_size;
        error = hipArrayGetInfo(nullptr, &level_to_size, nullptr, level_to);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to get last level info with code " << error << std::endl;
        }

        uint32_t width = m_width >> level;
        uint32_t height = m_height >> level;

        hipTextureObject_t texture_input;
        hipResourceDesc texture_resource;
        memset(&texture_resource, 0, sizeof(hipResourceDesc));

        texture_resource.resType            = hipResourceTypeArray;
        texture_resource.res.array.array    = level_from;

        hipTextureDesc texture_desc;
        memset(&texture_desc, 0, sizeof(hipTextureDesc));

        texture_desc.normalizedCoords   = 1;
        texture_desc.filterMode         = hipFilterModeLinear;
        texture_desc.addressMode[0]     = hipAddressModeClamp;
        texture_desc.addressMode[1]     = hipAddressModeClamp;
        texture_desc.addressMode[2]     = hipAddressModeClamp;
        texture_desc.readMode           = hipReadModeElementType;

        error = hipCreateTextureObject(&texture_input, &texture_resource, &texture_desc, nullptr);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to generate temporary texture object for mipmap level " << level - 1 << " with code " << error << std::endl;
        }

        hipSurfaceObject_t surface_output;
        hipResourceDesc surf_resource;
        memset(&surf_resource, 0, sizeof(hipResourceDesc));

        surf_resource.resType            = hipResourceTypeArray;
        surf_resource.res.array.array    = level_to;
        error = hipCreateSurfaceObject(&surface_output, &surf_resource);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to generate temporary surface object for mipmap level " << level << " with code " << error << std::endl;
        }

        dim3 block_dim(16, 16);
        dim3 grid_dim((width + block_dim.x - 1) / 16, (height + block_dim.y - 1) / 16);
        textureDownsampling<<<grid_dim, block_dim>>>(surface_output, texture_input, width, height);

        hipDeviceSynchronize();
        error = hipGetLastError();
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to launch kernel to generate mipmap for level " << level << " with code " << error << std::endl;
        }

        error = hipDestroyTextureObject(texture_input);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to delete temporary texture object for mipmap level " << level - 1 << " with code " << error << std::endl;
        }
        error = hipDestroySurfaceObject(surface_output);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to delete temporary surface object for mipmap level " << level << " with code " << error << std::endl;
        }

    }

    resource_desc.resType               = hipResourceTypeMipmappedArray;
    resource_desc.res.mipmap.mipmap     = m_d_mipmap_array;

    memset(&m_tex_description, 0, sizeof(m_tex_description));
    m_tex_description.normalizedCoords      = 1;
    m_tex_description.filterMode            = hipFilterModeLinear;
    m_tex_description.addressMode[0]        = hipAddressModeClamp;
    m_tex_description.addressMode[1]        = hipAddressModeClamp;
    m_tex_description.addressMode[2]        = hipAddressModeClamp;
    m_tex_description.maxMipmapLevelClamp   = num_levels - 1;
    m_tex_description.readMode              = hipReadModeElementType;

    m_tex_obj = 0;

    error = hipCreateTextureObject(&m_tex_obj, &resource_desc, &m_tex_description, nullptr);
    if ( error != hipSuccess ) {
        std::cerr << "Error: Failed to copy image to create texture object with code " << error << std::endl;
    }
}

Texture::~Texture() {
    hipError_t error;

    if ( m_d_mipmap_array ) {
        error = hipFreeMipmappedArray(m_d_mipmap_array);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to delete cudaMipmapedArray with code " << error << std::endl;
        }
    }
    if ( m_tex_obj ) {
        error = hipDestroyTextureObject(m_tex_obj);
        if ( error != hipSuccess ) {
            std::cerr << "Error: Failed to delete texture object with code " << error << std::endl;
        }
    }
}
